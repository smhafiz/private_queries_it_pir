// This file is part of BarrettCUDA v0.1.
// 
// BarrettCUDA is a fast(ish) implementation of finite field sparse
// matrix-vector multiplication (SpMV) for Nvidia GPU devices, written
// in CUDA C++. BarrettCUDA supports SpMV for matrices expressed in
// the 'compressed column storage' (CCS) sparse matrix representation
// over (i) the field of integers modulo an arbitrary multi-precision
// prime, or (ii) either of the binary fields GF(2^8) or GF(2^16).
// 
// Copyright (C) 2016, Ryan Henry and Syed Mahbub Hafiz.
// 
// BarrettCUDA is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published
// by the Free Software Foundation, either version 3 of the License,
// or (at your option) any later version.
// 
// BarrettCUDA is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with BarrettCUDA. If not, see <http://www.gnu.org/licenses/>.

#include <atomic>
#include <chrono>
#include <ratio>
#include <fstream>

#include "barrett.h"
#include "uintX.h"

#define DEBUG true

#define THREADS_PER_BLOCK(n) (n >= 512 ? 512 : n)
//#define THREADS_PER_BLOCK(n) (n >= 32 ? 32 : n)
#define NUM_BLOCKS(n) ((n + THREADS_PER_BLOCK(n) - 1) / THREADS_PER_BLOCK(n))

NTL_CLIENT

// specialization for uintX
template <typename T> struct _SpMV_specializer<T,0>
{
    static __device__ void device_SpMV(T * response, const T * query,
	const uint nvals, const T * vals, const uint ncols, const uint * cols,
	const uint * rows)
    {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= ncols) return;

	uintXp<T> res_lo = { 0 };
	T res_hi = { 0 };
	uint overflow = { 0 };

	// do the SpMV
	for (int j = cols[i]; j < cols[i+1]; ++j)
	{
	    mad(res_lo.lo, res_hi, overflow, vals[j], query[rows[j]]);
	}

	T * subtrahends = (T *)d_subtrahends;
	uintXp<T> * mu = (uintXp<T> *)d_mu;
	T * modulus = (T *)d_modulus;

	// do the Barrett reduction
	normalize(res_lo.lo, res_hi, subtrahends[overflow], (overflow ? -1: 0));
	uintXp<T> q = get_q(res_lo.lo, res_hi, *mu);
	uintXp<T> r2 = get_r2(q, *modulus);
	res_lo.hi = sub(res_lo.lo, res_hi, r2);
	if (res_lo.hi) sub_modulus(res_lo, *modulus);
	if (res_lo.hi) sub_modulus(res_lo, *modulus);

	// write final result to global memory
	response[i] = res_lo.lo;
    }
};

// specialization for GF28_Element
template <> struct _SpMV_specializer<GF28_Element,0>
{
    static __device__ void device_SpMV(GF28_Element * response,
	const GF28_Element * query, const uint nvals, const GF28_Element * vals,
	const uint ncols, const uint * cols, const uint * rows)
    {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= ncols) return;

	GF28_Element res = 0;
	for (int j = cols[i]; j < cols[i+1]; ++j)
	{
	    res ^= d_GF28_mult_table[vals[j]][query[rows[j]]];
	}
	response[i] = res;
    }
};

// specialization for GF216_Element
template <> struct _SpMV_specializer<GF216_Element,0>
{
    static __device__ void device_SpMV(GF216_Element * response,
	const GF216_Element * query, const uint nvals,
	const GF216_Element * vals, const uint ncols, const uint * cols,
	const uint * rows)
    {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= ncols) return;

	GF216_Element res = 0;
	for (int j = cols[i]; j < cols[i+1]; ++j)
	{
	    GF216_Element log_x = d_GF216_log_table[vals[j]];
	    GF216_Element log_y = d_GF216_log_table[query[rows[j]]];
	    res ^= d_GF216_exp_table[log_x+log_y];
	}
	response[i] = res;
    }
};

template <typename T>
void SpMV_ntl(NTL::vec_ZZ_p & response, const T * query, 
    const SparseMatrix<T> & matrix)
{
    for (int i = 0; i < matrix.ncols; i++)
    {
	response[i] = NTL::to_ZZ_p(0);
	for (int j = matrix.l_cols[i]; j < matrix.l_cols[i+1]; ++j)
	{
	    response[i] += to_ZZ_p(matrix.l_vals[j])
			 * to_ZZ_p(query[matrix.l_rows[j]]);
	}
    }
    //created by Warren
    //std::cout<<"SpMV_ntl output: \n";
    //std::cout<<response<<"\n";
}

#ifdef DEBUG
    template <typename T>
    void SpMV_ntl_barrett(NTL::vec_ZZ_p & response, const T * query,
	const SparseMatrix<T> & matrix, struct BarrettParams<T> & barrett)
    {
	NTL::vec_ZZ response_ZZ(INIT_SIZE, matrix.ncols);
	for (int i = 0; i < matrix.ncols; i++)
	{
	    response_ZZ[i] = NTL::to_ZZ(0);

	    for (int j = matrix.l_cols[i]; j < matrix.l_cols[i+1]; ++j)
	    {
		response_ZZ[i] += to_ZZ(matrix.l_vals[j]) * to_ZZ(query[matrix.l_rows[j]]);
	    }
	    uint overflow = (uint)NTL::trunc_long(response_ZZ[i] >> 2*BITS_IN(LIMBS_PER_UINTX), BITS_IN(sizeof(uint)));
	    response_ZZ[i] -= barrett.l_subtrahends[overflow];
	    NTL::ZZ q1 = response_ZZ[i] >> BITS_IN(LIMBS_PER_UINTX-1);
	    NTL::ZZ q2 = q1 * barrett.l_mu;
	    NTL::ZZ q3 = q2 >> BITS_IN(LIMBS_PER_UINTX+1);
	    NTL::ZZ r1 = response_ZZ[i] % NTL::power2_ZZ(BITS_IN(LIMBS_PER_UINTX+1));
	    NTL::ZZ r2 = q3 * barrett.l_modulus % NTL::power2_ZZ(BITS_IN(LIMBS_PER_UINTX+1));
	    NTL::ZZ r = (r1 - r2) % NTL::power2_ZZ(BITS_IN(LIMBS_PER_UINTX+1));
	    response[i] = NTL::to_ZZ_p(r);
	}
    //created by Warren
    //std::cout<<"query in SpMV_ntl_barrett: \n";
    //for(int i = 0; i <4; i++){std::cout<<to_ZZ(query[i])<<"\n";}
    //std::cout<< "Response in SpMV_ntl_barrett: \n";
    //std::cout<< response <<"\n";
    }
#endif // DEBUG

template <typename T>
void SpMV(T * l_response, const T * l_query,
	T * d_response, T * d_query, const hipStream_t & stream,
	const SparseMatrix<T> & matrix)
{
    gpuErrchk(hipMemcpyAsync(d_query, l_query, matrix.nrows * sizeof(T),
	hipMemcpyHostToDevice, stream));

    const dim3 Dg(NUM_BLOCKS(matrix.ncols), 1, 1);
    const dim3 Db(THREADS_PER_BLOCK(matrix.ncols), 1, 1);
    const size_t Ns = 0;

    SpMV_kernel<T> <<< Dg, Db, Ns, stream >>> (d_response, d_query,
	matrix.nvals, matrix.d_vals, matrix.ncols, matrix.d_cols, matrix.d_rows);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpyAsync(l_response, d_response, matrix.ncols * sizeof(T),
	hipMemcpyDeviceToHost, stream));
}

int main(int argc, char ** argv)
{
    int nstreams = 4;

    if (argc < 4)
    {
	std::cout << "Usage: " << argv[0] << " VALUES ROWS COLS\n\n";
	return 1;
    }

    time_t t0 = time(0);
    t0 = 1481878728;
    //NTL::SetSeed(to_ZZ(t0));
    //std::cout << "seed: " << t0 << "\n";

    struct SparseMatrix<uintX> matrix = { 0 };
    NTL::ZZ modulus;

    uint max_overflow;
    initMatrix(argv[1], argv[2], argv[3], modulus, matrix, max_overflow);
    NTL::ZZ_p::init(modulus);
    //std::cout<<"Line 201"<<std::endl;
    struct BarrettParams<uintX> barrett;
    initBarrett<uintX>(modulus, barrett, max_overflow);

    uintX * l_query, * d_query;
    gpuErrchk(hipHostMalloc((void**)&l_query,
	nstreams * matrix.nrows * sizeof(uintX)));
    gpuErrchk(hipMalloc((void**)&d_query,
	nstreams * matrix.nrows * sizeof(uintX)));

    uintX * l_response, * d_response;
    gpuErrchk(hipHostMalloc((void**)&l_response,
	nstreams * matrix.ncols * sizeof(uintX)));
    gpuErrchk(hipMalloc((void**)&d_response,
	nstreams * matrix.ncols * sizeof(uintX)));

    hipStream_t * streams = new hipStream_t[nstreams];
    for (int i = 0; i < nstreams; ++i) hipStreamCreate(&streams[i]);

    NTL::vec_vec_ZZ_p responses(INIT_SIZE, nstreams,
	NTL::vec_ZZ_p(INIT_SIZE, matrix.ncols));

    for (int i = 0; i < nstreams * matrix.nrows; i++)
    {
    NTL::ZZ_p temp = NTL::random_ZZ_p();
    //NTL::ZZ_p temp = to_ZZ_p(0); // inserted by Warren
    //std::cout<<temp<<"\n";
	to_uint<uintX>(temp, l_query[i]);
    }
    //to_uint<uintX>(to_ZZ_p(1), l_query[3]); // by Warren
     
    /*
    //input by Warren
    std::cout<<modulus<<"\n";
    std::cout<<"nrows: " << matrix.nrows<<"\n";
    std::cout<<"uintX size: " << sizeof(uintX)<<"\n";
    std::cout<<"l_query size: " << sizeof(l_query)<<"\n";
    std::cout<<"LIMBS_PER_UINTX: "<<LIMBS_PER_UINTX<<"\n";
    //std::cout << "Matrix: " << matrix. << "\n";
    std::cout << "l_query w/o p: \n";
    
    for(int i = 0;i< matrix.nrows;i++){
        //print_limbs<uintX>(l_query[i], LIMBS_PER_UINTX);
        std::cout<<to_ZZ(l_query[i])<<"\n";
    } 
    std::cout << "l_query in p: \n";
    
    for(int i = 0;i< matrix.nrows;i++){
        //print_limbs<uintX>(l_query[i], LIMBS_PER_UINTX);
        std::cout<<to_ZZ_p(l_query[i])<<"\n";
    }
    
    //std::cout<<to_string(l_query).name()<<"\n";
    //std::cout<<strcat(l_query)<<"\n";
    std::cout<<"\n";
    */
    //end Warren print

    std::atomic<int> cnt = ATOMIC_VAR_INIT(0);
    auto start = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds onesec{1000000000};
    //std::cout<<"Line 231"<<std::endl;
    while (std::chrono::duration_cast<std::chrono::duration<int,std::nano>>(std::chrono::high_resolution_clock::now() - start) < onesec)
    {
    // by Warren: commented line below to solve CPU problem uncomment to use GPU mode
	//#pragma omp parallel
	for (int i = 0; i < nstreams; i++)
	{
	    uintX * __l_response = l_response + i * matrix.ncols;
	    uintX * __d_response = d_response + i * matrix.ncols;
	    uintX * __l_query = l_query + i * matrix.nrows;
	    uintX * __d_query = d_query + i * matrix.nrows;		

	    SpMV<uintX>(__l_response, __l_query, __d_response,
		__d_query, streams[i], matrix);
 
	    //SpMV_ntl(responses[i], __l_query, matrix);
        
	    //SpMV_ntl_barrett(responses[i], __l_query, matrix, barrett);
       
        
	    std::atomic_fetch_add(&cnt, 1);
        
//	    for (int j = 0; j < matrix.nrows; j++)
//	    {
//		to_uint<uintX>(NTL::rep(NTL::random_ZZ_p()), __l_query[j]);
//	    }
	}
    }
    // Output
    //std::cout <<"Completed " << cnt << " SpMV per second\n";
    std::cout << cnt<<"\n"; // output for bash file data

    /*
    //Input by Warren
    std::cout << "l_response w/0 p: \n";
    for(int i = 0;i< 8;i++){
        //print_limbs<uintX>(l_response[i], LIMBS_PER_UINTX);
        std::cout<<to_ZZ_p(l_response[i])<<"\n";
    }
    */
    // end Warren Print

    // cleanup
    for (int i = 0; i < nstreams; ++i) gpuErrchk(hipStreamDestroy(streams[i]));
    delete [] streams;
    responses.kill();

    gpuErrchk(hipHostFree(l_query));
    gpuErrchk(hipFree(d_query));
    gpuErrchk(hipHostFree(l_response));
    gpuErrchk(hipFree(d_response));

    freeBarrett<uintX>(barrett);
    freeMatrix<uintX>(matrix);

    return 0;
}


template <typename T>
void initMatrix(const char * valfile, const char * rowfile,
	const char * colfile, NTL::ZZ & modulus,
	struct SparseMatrix<T> & matrix, uint & max_overflow)
{
    std::ifstream valstream(valfile, std::ifstream::in);
    if (!valstream) { cerr << "Error: opening VALS files\n"; exit(-1); }
    std::ifstream rowstream(rowfile, std::ifstream::in);
    if (!rowstream) { cerr << "Error: opening ROWS files\n"; exit(-1); }
    std::ifstream colstream(colfile, std::ifstream::in);
    if (!colstream) { cerr << "Error: opening COLS files\n"; exit(-1); }
    NTL::ZZ tmp_zz;

    valstream >> tmp_zz;

    modulus = NTL::trunc_ZZ(tmp_zz,sizeof(T)*8);

    rowstream >> matrix.nrows;
    rowstream >> matrix.nvals;
    matrix.l_rows = (uint *)malloc(matrix.nvals * sizeof(uint));
    gpuErrchk(hipMalloc((void**)&matrix.d_rows, matrix.nvals * sizeof(uint)));
    matrix.l_vals = (T *)malloc(matrix.nvals * sizeof(T));
    gpuErrchk(hipMalloc((void**)&matrix.d_vals, matrix.nvals * sizeof(T)));

    colstream >> matrix.ncols;
    matrix.l_cols = (uint *)malloc((matrix.ncols+1) * sizeof(uint));
    gpuErrchk(hipMalloc((void**)&matrix.d_cols,
	(matrix.ncols+1) * sizeof(uint)));

    NTL::ZZ_pPush p(modulus);
    for (int i = 0; i < matrix.nvals; i++)
    {
	NTL::ZZ_p tmp;
	valstream >> tmp;
	to_uint<T>(NTL::rep(tmp), matrix.l_vals[i]);
	rowstream >> matrix.l_rows[i];
    }
    valstream.close();
    rowstream.close();
    gpuErrchk(hipMemcpy(matrix.d_vals, matrix.l_vals, matrix.nvals * sizeof(T),
	hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(matrix.d_rows, matrix.l_rows,
	matrix.nvals * sizeof(uint), hipMemcpyHostToDevice));

    for (int i = 0; i < matrix.ncols+1; i++)
    {
	colstream >> matrix.l_cols[i];
    }
    colstream.close();

    NTL::ZZ max_col = NTL::to_ZZ(0);
    for (int i = 0; i < matrix.ncols; i++)
    {
	NTL::ZZ this_col = NTL::to_ZZ(0);
	for (int j = matrix.l_cols[i]; j < matrix.l_cols[i+1]; j++)
	{
	    this_col += to_ZZ(matrix.l_vals[j]);
	}
	max_col = (max_col > this_col) ? max_col : this_col;
    }
    max_col *= (modulus-1);
    max_col >>= (2*BITS_PER_LIMB*LIMBS_PER_UINTX);
    max_overflow = (uint)trunc_long(max_col, 32);

    gpuErrchk(hipMemcpy(matrix.d_cols, matrix.l_cols,
	(matrix.ncols+1) * sizeof(uint), hipMemcpyHostToDevice));
}

template <typename T>
void freeMatrix(struct SparseMatrix<T> & matrix)
{
    free(matrix.l_vals);
    free(matrix.l_rows);
    free(matrix.l_cols);
    gpuErrchk(hipFree(matrix.d_vals));
    gpuErrchk(hipFree(matrix.d_cols));
    gpuErrchk(hipFree(matrix.d_rows));
}

template <typename T>
void initBarrett(const NTL::ZZ & modulus_zz, BarrettParams<T> & barrett,
	const uint max_overflow)
{
    barrett.l_modulus = modulus_zz;
    barrett.l_mu = NTL::power2_ZZ(2*BITS_PER_LIMB*LIMBS_PER_UINTX) / modulus_zz;
    T modulus;
    to_uint<T>(modulus_zz, modulus);
    uintXp<T> mu;
    to_uint<uintXp<T>>(barrett.l_mu, mu);

    barrett.l_subtrahends.SetLength(max_overflow+1);
    T * subtrahends = (T *)malloc((max_overflow+1) * sizeof(T));
    for (int i = 0; i <= max_overflow; ++i)
    {
	barrett.l_subtrahends[i] = ((NTL::to_ZZ(i)
	    << (2*BITS_PER_LIMB*LIMBS_PER_UINTX)) / modulus_zz) * modulus_zz;
	NTL::BytesFromZZ((unsigned char *)&subtrahends[i],
	    barrett.l_subtrahends[i], LIMBS_PER_UINTX * sizeof(uint));
    }

    gpuErrchk(hipMalloc((void**)&barrett.d_modulus, sizeof(T)));
    gpuErrchk(hipMemcpy(barrett.d_modulus, &modulus, sizeof(T),
	hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_modulus), &barrett.d_modulus, sizeof(T *)));

    gpuErrchk(hipMalloc((void**)&barrett.d_mu, sizeof(uintXp<T>)));
    gpuErrchk(hipMemcpy(barrett.d_mu, &mu, sizeof(uintXp<T>),
	hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_mu), &barrett.d_mu, sizeof(uintXp<T> *)));

    gpuErrchk(hipMalloc((void**)&barrett.d_subtrahends,
	(max_overflow+1) * sizeof(T)));
    gpuErrchk(hipMemcpy(barrett.d_subtrahends, subtrahends,
	(max_overflow+1) * sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_subtrahends), &barrett.d_subtrahends,
	sizeof(T *)));

    free(subtrahends);
}

template<typename T>
void freeBarrett(struct BarrettParams<T> & barrett)
{
    barrett.l_subtrahends.kill();
    barrett.l_modulus.kill();
    barrett.l_mu.kill();
    gpuErrchk(hipFree(barrett.d_modulus));
    gpuErrchk(hipFree(barrett.d_mu));
    gpuErrchk(hipFree(barrett.d_subtrahends));
}
